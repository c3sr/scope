#include "hip/hip_runtime.h"

#include <benchmark/benchmark.h>

#include <iostream>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <hip/hip_runtime.h>

#include "init/init.hpp"
#include "utils/utils.hpp"
#include "vectoradd/args.hpp"

template <typename T, int COARSINING_FACTOR = 1, int BLOCK_SIZE = 1>
__global__ void cuda_vector_add(T *in1, T *in2, T *out, size_t len) {
  // todo: implement COARSINING_FACTOR
  int index = threadIdx.x + blockIdx.x * BLOCK_SIZE;
  if (index < len) {
    out[index] = in1[index] + in2[index];
  }
}

template <typename T, int COARSINING_FACTOR = 1, int BLOCK_SIZE = 128>
static void CUDA_VECTOR_ADD(benchmark::State &state) {
  if (!has_cuda) {
    state.SkipWithError("CUDA/VECTOR_ADD/BASIC no CUDA device found");
    return;
  }

  const size_t N = state.range(0);

  const dim3 blockDim(BLOCK_SIZE);
  const dim3 gridDim(ceil(((float) N) / blockDim.x));

  if (gridDim.x >= cuda_device_prop.maxGridSize[0]) {
    const auto str = fmt::format("CUDA/VECTOR_ADD/BASIC the grid dimension {} exceeds the max grid dimensions {}",
                                 gridDim.x, cuda_device_prop.maxGridSize[0]);
    state.SkipWithError(str.c_str());
    return;
  }

  if (gridDim.x >= CUDA_MAX_GRID_SIZE) {
    const auto str = fmt::format("CUDA/VECTOR_ADD/BASIC the grid dimension {} exceeds the max grid dimensions {}",
                                 gridDim.x, CUDA_MAX_GRID_SIZE);
    state.SkipWithError(str.c_str());
    return;
  }

  auto a = std::vector<T>(N);
  auto b = std::vector<T>(N);
  auto c = std::vector<T>(N);

  std::fill(a.begin(), a.end(), 1);
  std::fill(b.begin(), b.end(), 1);
  std::fill(c.begin(), c.end(), 0);

  T *d_a{nullptr}, *d_b{nullptr}, *d_c{nullptr};

  if (PRINT_IF_ERROR(hipMalloc((void **) &d_a, a.size() * sizeof(*a.data())))) {
    LOG(critical, "CUDA/VECTOR_ADD/BASIC device memory allocation failed for vector A");
    state.SkipWithError("CUDA/VECTOR_ADD/BASIC device memory allocation failed for vector A");
    return;
  }
  defer(hipFree(d_a));

  if (PRINT_IF_ERROR(hipMalloc((void **) &d_b, b.size() * sizeof(*b.data())))) {
    LOG(critical, "CUDA/VECTOR_ADD/BASIC device memory allocation failed for vector B");
    state.SkipWithError("CUDA/VECTOR_ADD/BASIC device memory allocation failed for vector B");
    return;
  }
  defer(hipFree(d_b));

  if (PRINT_IF_ERROR(hipMalloc((void **) &d_c, c.size() * sizeof(*c.data())))) {
    LOG(critical, "CUDA/VECTOR_ADD/BASIC device memory allocation failed for vector C");
    state.SkipWithError("CUDA/VECTOR_ADD/BASIC device memory allocation failed for vector C");
    return;
  }
  defer(hipFree(d_c));

  if (PRINT_IF_ERROR(hipMemcpy(d_a, a.data(), a.size() * sizeof(*a.data()), hipMemcpyHostToDevice))) {
    state.SkipWithError("CUDA/VECTOR_ADD/BASIC device memory copy failed for vector A");
    return;
  }

  if (PRINT_IF_ERROR(hipMemcpy(d_b, b.data(), b.size() * sizeof(*b.data()), hipMemcpyHostToDevice))) {
    state.SkipWithError("CUDA/VECTOR_ADD/BASIC device memory copy failed for vector B");
    return;
  }

  if (PRINT_IF_ERROR(hipMemcpy(d_c, c.data(), c.size() * sizeof(*c.data()), hipMemcpyHostToDevice))) {
    state.SkipWithError("CUDA/VECTOR_ADD/BASIC device memory copy failed for vector C");
    return;
  }

#ifdef USE_CUDA_EVENTS
  hipEvent_t start, stop;
  PRINT_IF_ERROR(hipEventCreate(&start));
  PRINT_IF_ERROR(hipEventCreate(&stop));
#endif // USE_CUDA_EVENTS

  for (auto _ : state) {
#ifdef USE_CUDA_EVENTS
    hipEventRecord(start, NULL);
#endif // USE_CUDA_EVENTS

    cuda_vector_add<T, COARSINING_FACTOR, BLOCK_SIZE><<<gridDim, blockDim>>>(d_a, d_b, d_c, N);

#ifdef USE_CUDA_EVENTS
    hipEventRecord(stop, NULL);
    const auto cuda_err = hipEventSynchronize(stop);
#else  // USE_CUDA_EVENTS
    const auto cuda_err = hipDeviceSynchronize();
#endif // USE_CUDA_EVENTS

    state.PauseTiming();
    if (PRINT_IF_ERROR(cuda_err)) {
      state.SkipWithError("CUDA/VECTOR_ADD/BASIC failed to launch kernel");
      break;
    }
#ifdef USE_CUDA_EVENTS
    float msecTotal = 0.0f;
    if (PRINT_IF_ERROR(hipEventElapsedTime(&msecTotal, start, stop))) {
      state.SkipWithError("CUDA/VECTOR_ADD/BASIC failed to get elapsed time");
      break;
    }
    state.SetIterationTime(msecTotal / 1000);
#endif // USE_CUDA_EVENTS
    state.ResumeTiming();
  }

  state.counters.insert({{"N", N},
                         {"BLOCK_SIZE", BLOCK_SIZE},
                         {"Flops", {1.0 * N, benchmark::Counter::kAvgThreadsRate}},
                         {"COARSINING_FACTOR", COARSINING_FACTOR}});
  state.SetBytesProcessed(int64_t(state.iterations()) * N * sizeof(T));
  state.SetItemsProcessed(int64_t(state.iterations()) * N);
}

#ifdef USE_CUDA_EVENTS
#ifndef FAST_MODE
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 32)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 32)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 32)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 32)->ALL_ARGS()->UseManualTime();

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 64)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 64)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 64)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 64)->ALL_ARGS()->UseManualTime();

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 128)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 128)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 128)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 128)->ALL_ARGS()->UseManualTime();

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 256)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 256)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 256)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 256)->ALL_ARGS()->UseManualTime();
#endif // FAST_MODE

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 512)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 512)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 512)->ALL_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 512)->ALL_ARGS()->UseManualTime();

#else // USE_CUDA_EVENTS
#ifndef FAST_MODE
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 32)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 32)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 32)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 32)->ALL_ARGS();

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 64)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 64)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 64)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 64)->ALL_ARGS();

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 128)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 128)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 128)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 128)->ALL_ARGS();

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 256)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 256)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 256)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 256)->ALL_ARGS();

#endif // FAST_MODE

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 512)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 512)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 512)->ALL_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 512)->ALL_ARGS();

#endif // USE_CUDA_EVENTS
