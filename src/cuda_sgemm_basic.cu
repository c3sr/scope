
#include <hip/hip_runtime.h>

__global__ void basic_matrix_multiply(float *A, float *B, float *C,
                                      int numARows, int numAColumns,
                                      int numBRows, int numBColumns) {
  //@@ Insert code to implement matrix multiplication here
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < numARows && col < numBColumns) {
    float sum = 0;
    for (int ii = 0; ii < numAColumns; ii++) {
      sum += A[row * numAColumns + ii] * B[ii * numBColumns + col];
    }
    C[row * numBColumns + col] = sum;
  }
}
