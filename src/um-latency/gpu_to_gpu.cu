#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <numa.h>

#include "init/init.hpp"
#include "utils/utils.hpp"

#include "um-latency/args.hpp"

#define NAME "UM/Latency/GPUToGPU"

template <bool NOOP = false>
__global__ void gpu_traverse(size_t *ptr, const size_t steps)
{

  if (NOOP)
  {
    return;
  }
  size_t next = 0;
  for (int i = 0; i < steps; ++i)
  {
    next = ptr[next];
  }
  ptr[next] = 1;
}

static void UM_Latency_GPUToGPU(benchmark::State &state) {

  if (!has_cuda) {
    state.SkipWithError(NAME " no CUDA device found");
    return;
  }

  if (!has_numa) {
    state.SkipWithError(NAME " NUMA not available");
    return;
  }

  const size_t steps = state.range(0);
  const int src_id = state.range(1);
  const int dst_id = state.range(2);

  const size_t stride = 65536 * 2;
  const size_t bytes = sizeof(size_t) * (steps + 1) * stride;


  if (PRINT_IF_ERROR(utils::cuda_reset_device(src_id))) {
    state.SkipWithError(NAME " failed to reset src device");
    return;
  }
  if (PRINT_IF_ERROR(utils::cuda_reset_device(src_id))) {
    state.SkipWithError(NAME " failed to reset dst device");
    return;
  }

  if (PRINT_IF_ERROR(hipSetDevice(dst_id))) {
    state.SkipWithError(NAME " failed to set CUDA dst device");
    return;
  }

  size_t *ptr = nullptr;
  if (PRINT_IF_ERROR(hipMallocManaged(&ptr, bytes))) {
    state.SkipWithError(NAME " failed to perform hipMallocManaged");
    return;
  }
  defer(hipFree(ptr));

  if (PRINT_IF_ERROR(hipMemset(ptr, 0, bytes))) {
    state.SkipWithError(NAME " failed to perform hipMemset");
    return;
  }
  // set up stride pattern
  for (size_t i = 0; i < steps; ++i)
  {
    ptr[i * stride] = (i + 1) * stride;
  }
  if (PRINT_IF_ERROR(hipSetDevice(src_id))) {
    state.SkipWithError(NAME " failed to set CUDA src device");
    return;
  }
  if (PRINT_IF_ERROR(hipDeviceSynchronize())) {
    state.SkipWithError(NAME " failed to synchronize");
    return;
  }

  if (PRINT_IF_ERROR(hipSetDevice(dst_id))) {
    state.SkipWithError(NAME " failed to set CUDA dst device");
    return;
  }
  hipEvent_t start, stop;
  if (PRINT_IF_ERROR(hipEventCreate(&start))) {
    state.SkipWithError(NAME " failed to create start event");
    return;
  }
  defer(hipEventDestroy(start));

  if (PRINT_IF_ERROR(hipEventCreate(&stop))) {
    state.SkipWithError(NAME " failed to create end event");
    return;
  }
  defer(hipEventDestroy(stop));



  for (auto _ : state) {
    if (PRINT_IF_ERROR(hipMemPrefetchAsync(ptr, bytes, src_id))) {
      state.SkipWithError(NAME " failed to prefetch to src");
      return;
    }
    if (PRINT_IF_ERROR(hipSetDevice(src_id))) {
      state.SkipWithError(NAME " failed to set CUDA src device");
      return;
    }
    if (PRINT_IF_ERROR(hipDeviceSynchronize())) {
      state.SkipWithError(NAME " failed to synchronize");
      return;
    }
    if (PRINT_IF_ERROR(hipSetDevice(dst_id))) {
      state.SkipWithError(NAME " failed to set dst device");
      return;
    }
    if (PRINT_IF_ERROR(hipDeviceSynchronize())) {
      state.SkipWithError(NAME " failed to synchronize");
      return;
    }

    hipEventRecord(start);
    gpu_traverse<<<1, 1>>>(ptr, steps);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float millis = 0;
    if (PRINT_IF_ERROR(hipEventElapsedTime(&millis, start, stop))) {
      state.SkipWithError(NAME " failed to get elapsed time");
      break;
    }
    state.SetIterationTime(millis / 1000);

  }

}

BENCHMARK(UM_Latency_GPUToGPU)->Apply(ArgsCountGpuGpuNoSelf)->UseManualTime();
