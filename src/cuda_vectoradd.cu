#include "hip/hip_runtime.h"

#include <benchmark/benchmark.h>

#include <iostream>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <hip/hip_runtime.h>

#include "init.hpp"
#include "utils.hpp"
#include "utils_cuda.hpp"
#include "utils_vectoradd.hpp"

template <typename T, int COARSINING_FACTOR = 1, int BLOCK_SIZE = 1>
__global__ void cuda_vector_add(T *in1, T *in2, T *out, size_t len) {
  // todo: implement COARSINING_FACTOR
  int index = threadIdx.x + blockIdx.x * BLOCK_SIZE;
  if (index < len) {
    out[index] = in1[index] + in2[index];
  }
}

template <typename T, int COARSINING_FACTOR = 1, int BLOCK_SIZE = 128>
static void CUDA_VECTOR_ADD(benchmark::State &state) {

  const size_t N = state.range(0);

  auto a = std::vector<T>(N);
  auto b = std::vector<T>(N);
  auto c = std::vector<T>(N);

  std::fill(a.begin(), a.end(), 1);
  std::fill(b.begin(), b.end(), 1);
  std::fill(c.begin(), c.end(), 0);

  T *d_a{nullptr}, *d_b{nullptr}, *d_c{nullptr};

  auto cuda_err = hipMalloc((void **)&d_a, a.size() * sizeof(*a.data()));
  if (cuda_err != hipSuccess) {
    LOG(critical,
        "CUDA/VECTOR_ADD/BASIC device memory allocation failed for vector A");
    return;
  }
  defer(hipFree(d_a));

  cuda_err = hipMalloc((void **)&d_b, b.size() * sizeof(*b.data()));
  if (cuda_err != hipSuccess) {
    LOG(critical,
        "CUDA/VECTOR_ADD/BASIC device memory allocation failed for vector B");
    return;
  }
  defer(hipFree(d_b));

  cuda_err = hipMalloc((void **)&d_c, c.size() * sizeof(*c.data()));
  if (cuda_err != hipSuccess) {
    LOG(critical,
        "CUDA/VECTOR_ADD/BASIC device memory allocation failed for vector C");
    return;
  }
  defer(hipFree(d_c));

  cuda_err = CUDA_PERROR(hipMemcpy(d_a, a.data(), a.size() * sizeof(*a.data()),
                                    hipMemcpyHostToDevice));
  if (cuda_err != hipSuccess) {
    return;
  }

  cuda_err = CUDA_PERROR(hipMemcpy(d_b, b.data(), b.size() * sizeof(*b.data()),
                                    hipMemcpyHostToDevice));
  if (cuda_err != hipSuccess) {
    return;
  }

  cuda_err = CUDA_PERROR(hipMemcpy(d_c, c.data(), c.size() * sizeof(*c.data()),
                                    hipMemcpyHostToDevice));
  if (cuda_err != hipSuccess) {
    return;
  }

  dim3 blockDim(BLOCK_SIZE);
  dim3 gridDim(ceil(((float)N) / blockDim.x));

  hipEvent_t start, stop;
  CUDA_PERROR(hipEventCreate(&start));
  CUDA_PERROR(hipEventCreate(&stop));

  for (auto _ : state) {
    hipEventRecord(start, NULL);

    cuda_vector_add<T, COARSINING_FACTOR, BLOCK_SIZE>
        <<<gridDim, blockDim>>>(d_a, d_b, d_c, N);

    cuda_err = hipDeviceSynchronize();

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    state.PauseTiming();
    if (CUDA_PERROR(cuda_err) != hipSuccess) {
      break;
    }

    float msecTotal = 0.0f;
    if (cuda_err = CUDA_PERROR(hipEventElapsedTime(&msecTotal, start, stop))) {
      state.SkipWithError("CUDA/VECTOR_ADD/BASIC failed to get elapsed time");
    }
    state.SetIterationTime(msecTotal / 1000);
    state.ResumeTiming();
  }

  state.counters.insert({{"N", N},
                         {"BLOCK_SIZE", BLOCK_SIZE},
                         {"COARSINING_FACTOR", COARSINING_FACTOR}});
  state.SetBytesProcessed(int64_t(state.iterations()) * N);
}

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 32)
    ->VECTORADD_ARGS()
    ->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 32)
    ->VECTORADD_ARGS()
    ->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 32)
    ->VECTORADD_ARGS()
    ->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 32)
    ->VECTORADD_ARGS()
    ->UseManualTime();

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 64)
    ->VECTORADD_ARGS()
    ->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 64)
    ->VECTORADD_ARGS()
    ->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 64)
    ->VECTORADD_ARGS()
    ->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 64)
    ->VECTORADD_ARGS()
    ->UseManualTime();

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 128)
    ->VECTORADD_ARGS()
    ->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 128)
    ->VECTORADD_ARGS()
    ->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 128)
    ->VECTORADD_ARGS()
    ->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 128)
    ->VECTORADD_ARGS()
    ->UseManualTime();

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 32)->VECTORADD_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 32)->VECTORADD_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 32)->VECTORADD_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 32)->VECTORADD_ARGS();

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 64)->VECTORADD_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 64)->VECTORADD_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 64)->VECTORADD_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 64)->VECTORADD_ARGS();

BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, char, 1, 128)->VECTORADD_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, int, 1, 128)->VECTORADD_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, float, 1, 128)->VECTORADD_ARGS();
BENCHMARK_TEMPLATE(CUDA_VECTOR_ADD, double, 1, 128)->VECTORADD_ARGS();
