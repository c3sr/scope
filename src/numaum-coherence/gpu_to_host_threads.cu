#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <numa.h>

#include "init/init.hpp"
#include "utils/utils.hpp"

#include "numaum-coherence/args.hpp"

#define NAME "NUMAUM/Coherence/GPUToHostThreads"


template <bool NOOP = false>
__global__ void gpu_write(char *ptr, const size_t count, const size_t stride)
{
  if (NOOP)
  {
    return;
  }

  // global ID
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  // lane ID 0-31
  const size_t lx = gx & 31;
  // warp ID
  size_t wx = gx / 32;
  const size_t numWarps = (gridDim.x * blockDim.x + 32 - 1) / 32;

  if (0 == lx)
  {
    for (size_t i = wx * stride; i < count; i += numWarps * stride)
    {
      ptr[i] = 0;
    }
  }
}

static void NUMAUM_Coherence_GPUToHostThreads(benchmark::State &state) {

  if (!has_cuda) {
    state.SkipWithError(NAME " no CUDA device found");
    return;
  }

  if (!has_numa) {
    state.SkipWithError(NAME " NUMA not available");
    return;
  }

  const size_t pageSize = page_size();

  const size_t threads = state.range(0);
  const auto bytes = 1ULL << static_cast<size_t>(state.range(1));
  const int numa_id = state.range(2);
  const int cuda_id = state.range(3);

  omp_set_num_threads(threads);
  if (threads != omp_get_max_threads()) {
    state.SkipWithError(NAME " failed to set OMP threads");
    return;
  }

  omp_numa_bind_node(numa_id);

  if (PRINT_IF_ERROR(utils::cuda_reset_device(cuda_id))) {
    state.SkipWithError(NAME " failed to reset device");
    return;
  }
  if (PRINT_IF_ERROR(hipSetDevice(cuda_id))) {
    state.SkipWithError(NAME " failed to set CUDA device");
    return;
  }

  char *ptr = nullptr;
  if (PRINT_IF_ERROR(hipMallocManaged(&ptr, bytes))) {
    state.SkipWithError(NAME " failed to perform hipMallocManaged");
    return;
  }
  defer(hipFree(ptr));

  if (PRINT_IF_ERROR(hipMemset(ptr, 0, bytes))) {
    state.SkipWithError(NAME " failed to perform hipMemset");
    return;
  }

  for (auto _ : state) {

    state.PauseTiming();

    hipError_t err = hipMemPrefetchAsync(ptr, bytes, cuda_id);
    if (hipErrorInvalidDevice == err) {
      gpu_write<<<256,256>>>(ptr, bytes, pageSize);
    }
    if (PRINT_IF_ERROR(hipDeviceSynchronize())) {
      state.SkipWithError(NAME " failed to synchronize");
      return;
    }
    state.ResumeTiming();


  #pragma omp parallel for schedule(static)
    for (size_t i = 0; i < bytes; i += pageSize) {
      benchmark::DoNotOptimize(ptr[i] = 0);
    }
    benchmark::ClobberMemory();

  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes));
  state.counters.insert({{"bytes", bytes}});

  // reset to run on any node
  omp_numa_bind_node(-1);
}

BENCHMARK(NUMAUM_Coherence_GPUToHostThreads)->Apply(ArgsThreadsCountNumaGpu)->UseRealTime();
