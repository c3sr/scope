
#include <hip/hip_runtime.h>
#if CUDA_VERSION_MAJOR >= 8

#include <assert.h>
#include <iostream>
#include <stdio.h>
#include <string.h>

#include <cuda_runtime.h>
#include <numa.h>

#include "init/init.hpp"
#include "utils/utils.hpp"

#include "numaum-latency/args.hpp"

#define NAME "NUMAUM/Latency/GPUToHost"

template <bool NOOP = false>
void cpu_traverse(size_t *ptr, const size_t steps) {

  if (NOOP) {
    return;
  }
  size_t next = 0;
  for (size_t i = 0; i < steps; ++i) {
    next = ptr[next];
  }
  ptr[next] = 1;
}

static void NUMAUM_Latency_GPUToHost(benchmark::State &state) {

  if (!has_cuda) {
    state.SkipWithError(NAME " no CUDA device found");
    return;
  }

  if (!has_numa) {
    state.SkipWithError(NAME " NUMA not available");
    return;
  }

  const size_t steps = state.range(0);
  const int numa_id  = state.range(1);
  const int cuda_id  = state.range(2);

  const size_t stride = 65536 * 2;
  const size_t bytes  = sizeof(size_t) * (steps + 1) * stride;

  numa_bind_node(numa_id);
  if (PRINT_IF_ERROR(cudaSetDevice(cuda_id))) {
    state.SkipWithError(NAME " failed to set CUDA device");
    return;
  }
  if (PRINT_IF_ERROR(cudaDeviceReset())) {
    state.SkipWithError(NAME " failed to reset device");
    return;
  }

  size_t *ptr = nullptr;
  if (PRINT_IF_ERROR(cudaMallocManaged(&ptr, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMallocManaged");
    return;
  }
  defer(cudaFree(ptr));

  if (PRINT_IF_ERROR(cudaMemset(ptr, 0, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMemset");
    return;
  }

  // set up stride pattern
  for (size_t i = 0; i < steps; ++i) {
    ptr[i * stride] = (i + 1) * stride;
  }
  if (PRINT_IF_ERROR(cudaDeviceSynchronize())) {
    state.SkipWithError(NAME " failed to synchronize");
    return;
  }

  for (auto _ : state) {
    state.PauseTiming();
    // prefetch to source
    if (PRINT_IF_ERROR(cudaMemPrefetchAsync(ptr, bytes, cuda_id))) {
      state.SkipWithError(NAME " failed to synchronize");
      return;
    }
    if (PRINT_IF_ERROR(cudaDeviceSynchronize())) {
      state.SkipWithError(NAME " failed to synchronize");
      return;
    }
    state.ResumeTiming();

    cpu_traverse(ptr, steps);
  }
  state.counters["strides"] = steps;

  // reset to run on any node
  numa_bind_node(-1);
}

BENCHMARK(NUMAUM_Latency_GPUToHost)->Apply(ArgsCountNumaGpu)->MinTime(0.1);

#endif