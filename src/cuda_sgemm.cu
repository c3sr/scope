#include "hip/hip_runtime.h"

#include <benchmark/benchmark.h>

#include <iostream>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <hip/hip_runtime.h>

#include "fmt/format.h"

#include "init.hpp"
#include "utils.hpp"
#include "utils_cuda.hpp"
#include "utils_sgemm.hpp"

enum class CUDA_BLAS_IMPLEMENTATION : int { BASIC = 1, TILED = 2 };

static std::string CUDA_BLAS_IMPLEMENTATION_STRING(const CUDA_BLAS_IMPLEMENTATION impl) {
  switch (impl) {
    case CUDA_BLAS_IMPLEMENTATION::BASIC:
      return "BASIC";
    case CUDA_BLAS_IMPLEMENTATION::TILED:
      return "TILED";
    default:
      return "UNDEFINED";
  }
}

template <int TILE_WIDTH>
__global__ void cuda_basic_matrix_multiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows,
                                           int numBColumns, int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < numARows && col < numBColumns) {
    float sum = 0;
    for (int ii = 0; ii < numAColumns; ii++) {
      sum += A[row * numAColumns + ii] * B[ii * numBColumns + col];
    }
    C[row * numBColumns + col] = sum;
  }
}

template <int TILE_WIDTH>
__global__ void cuda_tiled_matrix_multiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows,
                                           int numBColumns, int numCRows, int numCColumns) {
  __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y, Row = by * TILE_WIDTH + ty,
      Col      = bx * TILE_WIDTH + tx;
  float Pvalue = 0;

  for (int m = 0; m < (numAColumns - 1) / TILE_WIDTH + 1; ++m) {
    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    if (Row < numARows && m * TILE_WIDTH + tx < numAColumns) {
      ds_M[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];
    } else {
      ds_M[ty][tx] = 0;
    }
    if (Col < numBColumns && m * TILE_WIDTH + ty < numBRows) {
      ds_N[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];
    } else {
      ds_N[ty][tx] = 0;
    }

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix

#pragma unroll
    for (int k = 0; k < TILE_WIDTH; ++k) {
      Pvalue += ds_M[ty][k] * ds_N[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }
  if (Row < numCRows && Col < numCColumns) {
    C[Row * numCColumns + Col] = Pvalue;
  }
}

template <CUDA_BLAS_IMPLEMENTATION IMPLEMENTATION, int TILE_WIDTH>
static void CUDA_SGEMM(benchmark::State &state) {

  const std::string IMPLEMNTATION_NAME = CUDA_BLAS_IMPLEMENTATION_STRING(IMPLEMENTATION);

  const auto M     = state.range(0);
  const auto N     = state.range(1);
  const auto K     = state.range(2);
  const auto alpha = 1.0f;
  const auto beta  = 0.0f;

  (void) alpha;
  (void) beta;

  const auto numARows    = M;
  const auto numAColumns = K;
  const auto numBRows    = K;
  const auto numBColumns = N;
  const auto numCRows    = M;
  const auto numCColumns = N;

  (void) numARows;
  (void) numAColumns;
  (void) numBRows;
  (void) numBColumns;
  (void) numCRows;
  (void) numCColumns;

  auto a = std::vector<float>(M * K);
  auto b = std::vector<float>(K * N);
  auto c = std::vector<float>(M * N);

  std::iota(a.begin(), a.end(), 1);
  std::iota(b.begin(), b.end(), 1);
  std::fill(c.begin(), c.end(), 0);

  float *d_a{nullptr}, *d_b{nullptr}, *d_c{nullptr};

  auto cuda_err = hipMalloc((void **) &d_a, a.size() * sizeof(*a.data()));
  if (cuda_err != hipSuccess) {
    LOG(critical, "CUDA/SGEMM/{} device memory allocation failed for matrix A", IMPLEMNTATION_NAME);
    return;
  }
  defer(hipFree(d_a));

  cuda_err = hipMalloc((void **) &d_b, b.size() * sizeof(*b.data()));
  if (cuda_err != hipSuccess) {
    LOG(critical, "CUDA/SGEMM/{} device memory allocation failed for matrix B", IMPLEMNTATION_NAME);
    return;
  }
  defer(hipFree(d_b));

  cuda_err = hipMalloc((void **) &d_c, c.size() * sizeof(*c.data()));
  if (cuda_err != hipSuccess) {
    LOG(critical, "CUDA/SGEMM/{} device memory allocation failed for matrix C", IMPLEMNTATION_NAME);
    return;
  }
  defer(hipFree(d_c));

  cuda_err = CUDA_PERROR(hipMemcpy(d_a, a.data(), a.size() * sizeof(*a.data()), hipMemcpyHostToDevice));
  if (cuda_err != hipSuccess) {
    return;
  }

  cuda_err = CUDA_PERROR(hipMemcpy(d_b, b.data(), b.size() * sizeof(*b.data()), hipMemcpyHostToDevice));
  if (cuda_err != hipSuccess) {
    return;
  }

  cuda_err = CUDA_PERROR(hipMemcpy(d_c, c.data(), c.size() * sizeof(*c.data()), hipMemcpyHostToDevice));
  if (cuda_err != hipSuccess) {
    return;
  }

  dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
  dim3 gridDim(ceil(((float) numBColumns) / blockDim.x), ceil(((float) numARows) / blockDim.y));

  hipEvent_t start, stop;
  CUDA_PERROR(hipEventCreate(&start));
  CUDA_PERROR(hipEventCreate(&stop));

  for (auto _ : state) {
    hipEventRecord(start, NULL);

    switch (IMPLEMENTATION) {
      case CUDA_BLAS_IMPLEMENTATION::BASIC:
        cuda_basic_matrix_multiply<TILE_WIDTH>
            <<<gridDim, blockDim>>>(d_a, d_b, d_c, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
        break;
      case CUDA_BLAS_IMPLEMENTATION::TILED:
        cuda_tiled_matrix_multiply<TILE_WIDTH>
            <<<gridDim, blockDim>>>(d_a, d_b, d_c, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
        break;
    }

    cuda_err = hipDeviceSynchronize();

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    state.PauseTiming();
    if (CUDA_PERROR(cuda_err) != hipSuccess) {
      break;
    }

    float msecTotal = 0.0f;
    if (cuda_err = CUDA_PERROR(hipEventElapsedTime(&msecTotal, start, stop))) {
      state.SkipWithError(fmt::format("CUDA/SGEMM/{} failed to get elapsed time", IMPLEMNTATION_NAME).c_str());
    }
    state.SetIterationTime(msecTotal / 1000);
    state.ResumeTiming();
  }

  state.counters.insert({{"M", M}, {"N", N}, {"K", K}});
  if (IMPLEMENTATION != CUDA_BLAS_IMPLEMENTATION::BASIC) {
    state.counters.insert({{"TILE_WIDTH", TILE_WIDTH}});
  }
  state.SetBytesProcessed(int64_t(state.iterations()) * 2 * M * N * K);
}

static void CUDA_SGEMM_BASIC(benchmark::State &state) {
  constexpr auto TILE_WIDTH = 16; // this is not used
  CUDA_SGEMM<CUDA_BLAS_IMPLEMENTATION::BASIC, TILE_WIDTH>(state);
}

template <int TILE_WIDTH>
static void CUDA_SGEMM_TILED(benchmark::State &state) {
  CUDA_SGEMM<CUDA_BLAS_IMPLEMENTATION::TILED, TILE_WIDTH>(state);
}

BENCHMARK(CUDA_SGEMM_BASIC)->SGEMM_ARGS()->UseManualTime();
BENCHMARK(CUDA_SGEMM_BASIC)->SGEMM_ARGS();

BENCHMARK_TEMPLATE(CUDA_SGEMM_TILED, 16)->SGEMM_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_SGEMM_TILED, 32)->SGEMM_ARGS()->UseManualTime();
BENCHMARK_TEMPLATE(CUDA_SGEMM_TILED, 64)->SGEMM_ARGS()->UseManualTime();

BENCHMARK_TEMPLATE(CUDA_SGEMM_TILED, 16)->SGEMM_ARGS();
BENCHMARK_TEMPLATE(CUDA_SGEMM_TILED, 32)->SGEMM_ARGS();
BENCHMARK_TEMPLATE(CUDA_SGEMM_TILED, 64)->SGEMM_ARGS();
