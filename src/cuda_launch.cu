#include "hip/hip_runtime.h"
#include <benchmark/benchmark.h>

#include <iostream>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <hip/hip_runtime.h>

#include "init.hpp"
#include "utils.hpp"
#include "utils_cuda.hpp"

enum class CUDA_LAUNCH_IMPLEMENTATION : int { EMPTY = 1, ADDTWO, RELU };

static inline std::string CUDA_LAUNCH_IMPLEMENTATION_STRING(const CUDA_LAUNCH_IMPLEMENTATION impl) {
  switch (impl) {
    case CUDA_LAUNCH_IMPLEMENTATION::EMPTY:
      return "EMPTY";
    case CUDA_LAUNCH_IMPLEMENTATION::ADDTWO:
      return "ADDTWO";
    case CUDA_LAUNCH_IMPLEMENTATION::RELU:
      return "RELU";
    default:
      return "UNDEFINED";
  }
}

template <typename T, int ITERATION_COUNT, int BLOCK_SIZE>
__global__ void cuda_empty_kernel(T *vec, size_t len) {
#pragma unroll
  for (int ii = 0; ii < ITERATION_COUNT; ii++) {
  }
}

template <typename T, int ITERATION_COUNT, int BLOCK_SIZE>
__global__ void cuda_add_two_kernel(T *vec, size_t len) {
  int index = threadIdx.x + blockIdx.x * BLOCK_SIZE;
  if (index < len) {
#pragma unroll
    for (int ii = 0; ii < ITERATION_COUNT; ii++) {
      vec[index] += 2;
    }
  }
}

template <typename T, int ITERATION_COUNT, int BLOCK_SIZE>
__global__ void cuda_relu_kernel(T *vec, size_t len) {
  int index = threadIdx.x + blockIdx.x * BLOCK_SIZE;
  if (index < len) {
#pragma unroll
    for (int ii = 0; ii < ITERATION_COUNT; ii++) {
      vec[index] = std::max(vec[index], 0);
    }
  }
}

template <CUDA_LAUNCH_IMPLEMENTATION IMPLEMENTATION, typename T, int LAUNCH_COUNT = 1, int ITERATION_COUNT = 1,
          int BLOCK_SIZE = 128>
static void CUDA_LAUNCH(benchmark::State &state) {

  const std::string IMPLEMNTATION_NAME = CUDA_LAUNCH_IMPLEMENTATION_STRING(IMPLEMENTATION);

  const size_t N = state.range(0);

  auto a = std::vector<T>(N);

  std::fill(a.begin(), a.end(), 1);

  T *d_a{nullptr};

  auto cuda_err = hipMalloc((void **) &d_a, a.size() * sizeof(*a.data()));
  if (cuda_err != hipSuccess) {
    LOG(critical, "CUDA/LAUNCH/{} device memory allocation failed for vector A", IMPLEMNTATION_NAME);
    return;
  }
  defer(hipFree(d_a));

  cuda_err = CUDA_PERROR(hipMemcpy(d_a, a.data(), a.size() * sizeof(*a.data()), hipMemcpyHostToDevice));
  if (cuda_err != hipSuccess) {
    return;
  }

  dim3 blockDim(BLOCK_SIZE);
  dim3 gridDim(ceil(((float) N) / blockDim.x));

  hipEvent_t start, stop;
  CUDA_PERROR(hipEventCreate(&start));
  CUDA_PERROR(hipEventCreate(&stop));

  for (auto _ : state) {
    hipEventRecord(start, NULL);

    for (const int ii = 0; ii < LAUNCH_COUNT; ii++) {
      switch (IMPLEMENTATION) {
        case CUDA_LAUNCH_IMPLEMENTATION::EMPTY:
          cuda_empty_kernel<T, ITERATION_COUNT, BLOCK_SIZE><<<gridDim, blockDim>>>(d_a, N);
          break;
        case CUDA_LAUNCH_IMPLEMENTATION::ADDTWO:
          cuda_add_two_kernel<T, ITERATION_COUNT, BLOCK_SIZE><<<gridDim, blockDim>>>(d_a, N);
          break;
        case CUDA_LAUNCH_IMPLEMENTATION::RELU:
          cuda_relu_kernel<T, ITERATION_COUNT, BLOCK_SIZE><<<gridDim, blockDim>>>(d_a, N);
          break;
      }
    }

    cuda_err = hipDeviceSynchronize();

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    state.PauseTiming();
    if (CUDA_PERROR(cuda_err) != hipSuccess) {
      break;
    }

    float msecTotal = 0.0f;
    if (cuda_err = CUDA_PERROR(hipEventElapsedTime(&msecTotal, start, stop))) {

      state.SkipWithError(fmt::format("CUDA/LAUNCH/{} failed to get elapsed time", IMPLEMNTATION_NAME).c_str());
    }
    state.SetIterationTime(msecTotal / 1000);
    state.ResumeTiming();
  }

  state.counters.insert(
      {{"N", N}, {"BLOCK_SIZE", BLOCK_SIZE}, {"ITERATION_COUNT", ITERATION_COUNT}, {"LAUNCH_COUNT", LAUNCH_COUNT}});
  state.SetBytesProcessed(int64_t(state.iterations()) * N);
}