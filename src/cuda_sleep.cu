#include "hip/hip_runtime.h"
#include <benchmark/benchmark.h>

#include <iostream>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <hip/hip_runtime.h>

#include "fmt/format.h"

#include "init.hpp"
#include "utils.hpp"
#include "utils_cuda.hpp"

using clock_value_t = long long;

// This is a kernel that does no real work but runs at least for a specified number of clocks
__device__ void sleep(clock_value_t sleep_cycles) {
  clock_value_t start = clock64();
  clock_value_t cycles_elapsed;

  // The code below should work like
  // this (thanks to modular arithmetics):
  //
  // clock_offset = (clock_t) (end_clock > start_clock ?
  //                           end_clock - start_clock :
  //                           end_clock + (0xffffffffu - start_clock));
  //
  // Indeed, let m = 2^32 then
  // end - start = end + m - start (mod m).
  do {
    cycles_elapsed = clock64() - start;
  } while (cycles_elapsed < sleep_cycles);
}
